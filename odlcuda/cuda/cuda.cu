#include "hip/hip_runtime.h"
#include <algorithm>
#include <memory>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

// thrust
#include <odl_cpp_utils/cuda/disableThrustWarnings.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/adjacent_difference.h>
#include <odl_cpp_utils/cuda/enableThrustWarnings.h>

// ODL
#include <odlcuda/cuda/DeviceVectorImpl.h>
#include <odlcuda/cuda/TypeMacro.h>
#include <odlcuda/cuda/CudaVectorImpl.h>

// Utils
#include <odl_cpp_utils/cuda/thrustUtils.h>
#include <odl_cpp_utils/utils/cast.h>
#include <odl_cpp_utils/cuda/cutil_math.h>
#include <odl_cpp_utils/cuda/errcheck.h>

__global__ void convKernel(const float* source, const float* kernel,
                           float* target, const int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= len) return;

    float value = 0.0f;

    for (int i = 0; i < len; i++) {
        value += source[i] *
                 kernel[(len + len / 2 + idx - i) % len]; // Positive modulo
    }

    target[idx] = value;
}

void convImpl(const DeviceVector<float>& source,
              const DeviceVector<float>& kernel, DeviceVector<float>& target) {
    size_t len = source.size();
    unsigned dimBlock = 256;
    unsigned dimGrid = narrow_cast<unsigned>(1 + (len / dimBlock));

    convKernel<<<dimGrid, dimBlock>>>(source.data(), kernel.data(), target.data(), narrow_cast<int>(len));
    CUDA_KERNEL_ERRCHECK;
}

__global__ void forwardDifferenceKernel(const int len, const float* source,
                                        float* target) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1;
         idx += blockDim.x * gridDim.x) {
        target[idx] = source[idx + 1] - source[idx];
    }
}
void forwardDifferenceImpl(const DeviceVector<float>& source,
                           DeviceVector<float>& target) {
    size_t len = source.size();
    unsigned dimBlock(256);
    unsigned dimGrid(
        std::min<unsigned>(128, narrow_cast<unsigned>(1 + (len / dimBlock))));

    forwardDifferenceKernel<<<dimBlock, dimGrid>>>(narrow_cast<int>(len), source.data(), target.data());
    CUDA_KERNEL_ERRCHECK;
}

__global__ void forwardDifferenceAdjointKernel(const int len,
                                               const float* source,
                                               float* target) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; idx < len - 1;
         idx += blockDim.x * gridDim.x) {
        target[idx] = -source[idx] + source[idx - 1];
    }
}
void forwardDifferenceAdjointImpl(const DeviceVector<float>& source,
                                  DeviceVector<float>& target) {
    size_t len = source.size();
    unsigned dimBlock(256);
    unsigned dimGrid(
        std::min<unsigned>(128u, narrow_cast<unsigned>(1 + (len / dimBlock))));

    forwardDifferenceAdjointKernel<<<dimBlock, dimGrid>>>(narrow_cast<int>(len), source.data(), target.data());
    CUDA_KERNEL_ERRCHECK;
}

void maxVectorVectorImpl(const DeviceVector<float>& v1,
                         const DeviceVector<float>& v2,
                         DeviceVector<float>& target) {
    thrust::transform(v1.begin(), v1.end(), v2.begin(), target.begin(),
                      thrust::maximum<float>());
}

void maxVectorScalarImpl(const DeviceVector<float>& source, float scalar,
                         DeviceVector<float>& target) {
    auto scalarIter = thrust::make_constant_iterator(scalar);
    thrust::transform(source.begin(), source.end(), scalarIter, target.begin(),
                      thrust::maximum<float>());
}

struct DivideFunctor {
    __host__ __device__ float operator()(const float& dividend,
                                         const float& divisor) {
        return divisor != 0.0f ? dividend / divisor : 0.0f;
    }
};
void divideVectorVectorImpl(const DeviceVector<float>& dividend,
                            const DeviceVector<float>& divisor,
                            DeviceVector<float>& quotient) {
    thrust::transform(dividend.begin(), dividend.end(), divisor.begin(),
                      quotient.begin(), DivideFunctor());
}

void addScalarImpl(const DeviceVector<float>& source, float scalar,
                   DeviceVector<float>& target) {
    auto scalarIter = thrust::make_constant_iterator(scalar);
    thrust::transform(source.begin(), source.end(), scalarIter, target.begin(),
                      thrust::plus<float>());
}

__global__ void forwardDifference2DKernel(const int cols, const int rows,
                                          const float* data, float* dx,
                                          float* dy) {
    for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < cols - 1;
         idy += blockDim.y * gridDim.y) {
        for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
             idx < rows - 1; idx += blockDim.x * gridDim.x) {
            const auto index = idx + rows * idy;

            dx[index] = data[index + 1] - data[index];
            dy[index] = data[index + rows] - data[index];
        }
    }
}

void forwardDifference2DImpl(const DeviceVector<float>& source,
                             DeviceVector<float>& dx, DeviceVector<float>& dy,
                             const int cols, const int rows) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(32, 32);

    forwardDifference2DKernel<<<dimGrid, dimBlock>>>(cols, rows, source.data(), dx.data(), dy.data());
    CUDA_KERNEL_ERRCHECK;
}

__global__ void forwardDifference2DAdjointKernel(const int cols, const int rows,
                                                 const float* dx,
                                                 const float* dy,
                                                 float* target) {
    for (auto idy = blockIdx.y * blockDim.y + threadIdx.y + 1; idy < cols - 1;
         idy += blockDim.y * gridDim.y) {
        for (auto idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
             idx < rows - 1; idx += blockDim.x * gridDim.x) {
            const auto index = idx + rows * idy;

            target[index] =
                -dx[index] + dx[index - 1] - dy[index] + dy[index - rows];
        }
    }
}

void forwardDifference2DAdjointImpl(const DeviceVector<float>& dx,
                                    const DeviceVector<float>& dy,
                                    DeviceVector<float>& target, const int cols,
                                    const int rows) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(32, 32);

    forwardDifference2DAdjointKernel<<<dimGrid, dimBlock>>>(cols, rows, dx.data(), dy.data(), target.data());
    CUDA_KERNEL_ERRCHECK;
}

#define PI 3.141592653589793f

__global__ void gaussianBlurXkernel(const float* source, float* target,
                                    const uint2 imageSize, float sigma_x,
                                    int kernel_width) {
    int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y);

    if (id.x >= imageSize.x || id.y >= imageSize.y) return;

    const float a = 1.0f / (sigma_x * sqrtf(2.0f * PI));
    const float b = 2.0f * sigma_x * sigma_x;

    float value = 0.0f;

    for (int i = id.x - kernel_width; i < id.x + kernel_width; i++) {
        int x = i;
        if (x < 0) x = -x;
        if (x >= imageSize.x) x = 2 * imageSize.x - x - 1;
        if (x < 0 || x >= imageSize.x) continue;

        float dx = static_cast<float>(i - id.x);
        float fx = a * expf(-dx * dx / b);
        int point_id = x + imageSize.x * id.y;
        value += fx * source[point_id];
    }

    unsigned idx = id.x + imageSize.x * id.y;
    target[idx] = value;
}

__global__ void gaussianBlurYkernel(const float* source, float* target,
                                    const uint2 imageSize, float sigma_y,
                                    int kernel_height) {
    int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y);

    if (id.x >= imageSize.x || id.y >= imageSize.y) return;

    const float a = 1.0f / (sigma_y * sqrtf(2.0f * PI));
    const float b = 2.0f * sigma_y * sigma_y;

    float value = 0.0f;

    for (int i = id.y - kernel_height; i < id.y + kernel_height; i++) {
        int y = i;
        if (y < 0) y = -y;
        if (y >= imageSize.y) y = 2 * imageSize.y - y - 1;
        if (y < 0 || y >= imageSize.y) continue;

        float dx = static_cast<float>(i - id.y);
        float fx = a * expf(-dx * dx / b);
        int point_id = id.x + imageSize.x * y;
        value += fx * source[point_id];
    }

    unsigned idx = id.x + imageSize.x * id.y;
    target[idx] = value;
}

void gaussianBlurImpl(const DeviceVector<float>& image,
                      DeviceVector<float>& temporary, DeviceVector<float>& out,
                      const int image_width, const int image_height,
                      const float sigma_x, const float sigma_y,
                      const int kernel_width, const int kernel_height) {
    dim3 dimBlock(16, 16);
    dim3 dimGrid(narrow_cast<unsigned>(1 + (image_width / dimBlock.x)),
                 narrow_cast<unsigned>(1 + (image_height / dimBlock.y)));

    gaussianBlurXkernel<<<dimGrid, dimBlock>>>(image.data(),
                                               temporary.data(),
                                               make_uint2(narrow_cast<unsigned>(image_width),
                                                          narrow_cast<unsigned>(image_height)),
                                               sigma_x, kernel_width);
    CUDA_KERNEL_ERRCHECK;
    gaussianBlurYkernel<<<dimGrid, dimBlock>>>(temporary.data(),
                                               out.data(),
                                               make_uint2(narrow_cast<unsigned>(image_width),
                                                          narrow_cast<unsigned>(image_height)),
                                               sigma_y,
                                               kernel_height);
    CUDA_KERNEL_ERRCHECK;
}
