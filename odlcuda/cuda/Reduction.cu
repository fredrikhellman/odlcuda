// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

// thrust
#include <odl_cpp_utils/cuda/disableThrustWarnings.h>
#include <thrust/transform.h>
#include <thrust/extrema.h>
#include <odl_cpp_utils/cuda/enableThrustWarnings.h>

#include <iostream>

// ODL
#include <odlcuda/cuda/Reduction.h>
#include <odlcuda/cuda/DeviceVectorImpl.h>
#include <odlcuda/cuda/CudaVectorImpl.h>

// Utils
#include <math.h>

// Instantiate the methods for each type
template <>
float reduction_sum(const CudaVectorImpl<float>& v) {
    return thrust::reduce(v._impl->begin(), v._impl->end());
}

template <>
float reduction_prod(const CudaVectorImpl<float>& v) {
    return thrust::reduce(v._impl->begin(), v._impl->end(), 1.0f, thrust::multiplies<float>());
}

template <>
float reduction_min(const CudaVectorImpl<float>& v) {
    return *thrust::min_element(v._impl->begin(), v._impl->end());
}

template <>
float reduction_max(const CudaVectorImpl<float>& v) {
    return *thrust::max_element(v._impl->begin(), v._impl->end());
}